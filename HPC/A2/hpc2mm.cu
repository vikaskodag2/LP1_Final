#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__global__ void kernel(float* da,float* db,float* dc,int n)
{
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int sum=0;

    for(int i=0;i<n;i++)
    {
        sum =sum + da[tx*n+i]*db[i*n+ty];
    }
    dc[tx*n+ty]=sum;
    
}
void init(float* a,int n)
{
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            a[i*n+j]=rand%n+1;
        }
    }
}

void printm(float *a,int n)
{
    for(int i=0;i<n;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf(" %f",a[i*n+j]);
        }
        printf("\n");
    }
}

int main()
{
float *a,*b,*c;
float *da,*db,*dc;

int n;
n=3;


a=(float*)malloc(sizeof(float)*n*n);
b=(float*)malloc(sizeof(float)*n*n);
c=(float*)malloc(sizeof(float)*n*n);

init(a,n);
init(b,n);

printm(a,n);
printm(b,n);


hipMalloc(&da,sizeof(float)*n*n);
hipMalloc(&dc,sizeof(float)*n*n);
hipMalloc(&db,sizeof(float)*n*n);

hipMemcpy(da,a,sizeof(float)*n*n,hipMemcpyHostToDevice);
hipMemcpy(db,b,sizeof(float)*n*n,hipMemcpyHostToDevice);

dim3 dimGrid(1,1);
dim3 dimBlock(n,n);


kernel<<<dimGrid,dimBlock>>>(da,db,dc,n);

hipMemcpy(c,dc,sizeof(float)*n*n,hipMemcpyDeviceToHost);
printm(c,n);

hipFree(da);
hipFree(db);
hipFree(dc);

delete[] a;
delete[] b;
delete[] c;	
	
	
return 0;
}
