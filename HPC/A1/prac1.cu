#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>

void init_array(float *a,int n);

__global__
void sum(float* input)
{
	int tid=threadIdx.x;
	int no_threads=blockDim.x;
	int step_size=1;
	
	while(no_threads>0)
	{
		 //printf("\n tid:%d  no_threads:%d  step_size:%d \n",tid,no_threads,step_size);
		 if(tid<no_threads)
		 {
		 	int fst=tid*step_size*2;
		 	int snd=fst+step_size;
		 	input[fst]+=input[snd];
		 }
		 step_size <<= 1;
		 no_threads >>=1;
	}
}

__global__
void max(float* input)
{
	int tid=threadIdx.x;
	int no_threads=blockDim.x;
	int step_size=1;
	
	while(no_threads>0)
	{
		if(tid<no_threads)
		{
			int fst=tid*step_size*2;
			int snd=fst+step_size;
			if(input[fst]<input[snd])
				input[fst]=input[snd];
		}
		step_size <<= 1;
		no_threads >>=1;
	}
}
 
__global__
void min(float* input)
{
	int tid=threadIdx.x;
	int no_threads=blockDim.x;
	int step_size=1;
	
	while(no_threads>0)
	{
		if(tid<no_threads)
		{
			int fst=tid*step_size*2;
			int snd=fst+step_size;
			if(input[fst]>input[snd])
				input[fst]=input[snd];
		}
		step_size <<= 1;
		no_threads >>=1;
	}
}

__global__
void std_(float* input,float avg)
{
	int tid=threadIdx.x;
	int no_threads=blockDim.x;
	int step_size=1;
	
	while(no_threads>0)
	{
		if(tid<no_threads)
		{
			int fst=tid*step_size*2;
			int snd=fst+step_size;
			if(step_size==1){
			input[fst] = (input[fst]-avg)*(input[fst]-avg);
      			input[snd] = (input[snd]-avg)*(input[snd]-avg);
      			input[fst] += input[snd];}
      			else{
      			input[fst] += input[snd];
      			}
		}
		step_size <<= 1;
		no_threads >>=1;
	}
}




int main()
{
	int n=4;
	float *a,*d_a;
	float SUM,MAX,MIN,STD_,avg;
	a=(float*)malloc(sizeof(float)*n);
	hipMalloc(&d_a,n*sizeof(float));
	init_array(a,n);
	for(int i=0;i<n;i++)
		printf("%f   ",a[i]);
	float m;
	
	
	
	
//	for(int i=0;i<n;i++)  //sequential sum
//		m=m+a[i];

/**********************************************************************************************/
	hipMemcpy(d_a,a,n*sizeof(float),hipMemcpyHostToDevice);
	sum<<<1,n/2>>>(d_a);
	hipMemcpy(&SUM,d_a,sizeof(float),hipMemcpyDeviceToHost);
	printf("SUM:%f",SUM);
/**********************************************************************************************/


/**********************************************************************************************/
	hipMemcpy(d_a,a,n*sizeof(float),hipMemcpyHostToDevice);
	max<<<1,n/2>>>(d_a);
	hipMemcpy(&MAX,d_a,sizeof(float),hipMemcpyDeviceToHost);
	printf("\nMax:%f",MAX);
/**********************************************************************************************/


/**********************************************************************************************/
	hipMemcpy(d_a,a,n*sizeof(float),hipMemcpyHostToDevice);
	min<<<1,n/2>>>(d_a);
	hipMemcpy(&MIN,d_a,sizeof(float),hipMemcpyDeviceToHost);
	printf("\nMin:%f",MIN);
/**********************************************************************************************/


/**********************************************************************************************/
	avg=SUM/n;
	hipMemcpy(d_a,a,n*sizeof(float),hipMemcpyHostToDevice);
	std_<<<1,n/2>>>(d_a,avg);
	hipMemcpy(&STD_,d_a,sizeof(float),hipMemcpyDeviceToHost);
	STD_ = STD_/n;
	STD_ = sqrt(STD_);
	printf("\nSTD:%f",STD_);
/**********************************************************************************************/

	hipFree(d_a);
	delete[] a;
	return 0;
}

void init_array(float*a,int n)
{
  for(int i=0;i<n;i++)
     a[i] = rand()%n + 1;
}
