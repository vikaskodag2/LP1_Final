
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void multiply(int *ad, int *bd, int *cd, int n)
{

	int row = blockIdx.x ;
	int col = blockIdx.y ;

	int sum = 0;

	for (int i = 0; i < n; i++)

	{
		sum = sum + ad[row * n + i] * bd[i * n + col];
	}

	cd[row * n + col] = sum;
}

int main()
{

	cout << "Enter the size" << endl;
	int n;

	cin >> n;

	int a[n * n], b[n * n], c[n * n];

	for (int i = 0; i < n; i++)
	{

		for (int j = 0; j < n; j++)

		{

			a[i * n + j] = i;
			b[i * n + j] = i;
		}
	}
for (int i = 0; i < n; i++)
	{

		for (int j = 0; j < n; j++)
		{

			cout << a[i * n + j] << " ";
		}
		cout << endl;
	}
	int size = n * n * sizeof(int);

	int *ad, *bd, *cd;

	hipEvent_t start, end;

	hipMalloc(&ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);

	hipMalloc(&bd, size);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	hipMalloc(&cd, size);

	dim3 grid(n, n, 1);
	dim3 block(1, 1, 1);

	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start);

	multiply<<<grid, block>>>(ad, bd, cd, n);

	hipEventRecord(end);
	hipEventSynchronize(end);

	float time = 0;

	hipEventElapsedTime(&time, start, end);

	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{

		for (int j = 0; j < n; j++)
		{

			cout << c[i * n + j] << " ";
		}
		cout << endl;
	}

	cout << "The time required is " << time << endl;
}
