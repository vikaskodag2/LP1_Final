#include<hip/hip_runtime.h>
#include<stdio.h>

int main(void) {
    void MatrixMultiplication(float *, float *, float *, int,int,int);
    const int k =5 ;
    const int m=4,n=3;

    float M[k*n], N[n*m], P[k*m];
    for(int i = 0; i < (k*n) ; i++) {
        M[i] = 6;

    }
    for(int i = 0; i < (n*m) ; i++) {
            //M[i] = 6;
            N[i] = 6;
           // P[i] = 0;
        }
    for(int i = 0; i < (k*m) ; i++) {
            //M[i] = 6;
           // N[i] = 6;
            P[i] = 0;
        }
    MatrixMultiplication(M, N, P,m,n,k);
    for(int i = 0; i < (k*m) ; i++) {
        printf("%f \n", P[i]);
    }
    int quit;
    scanf("%d",&quit);
    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int N,int M) {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    printf("%d %d\n",tx,ty);
    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

    for(int k = 0; k <N ; ++k) {
        float Mdelement = Md[tx*N + k];
        float Ndelement = Nd[k*M + ty];
        Pvalue += (Mdelement*Ndelement);
    }

    Pd[tx*M + ty] = Pvalue;
}

void MatrixMultiplication(float *M, float *N, float *P, int m,int n,int k) {
    //int size = Width*Width*sizeof(float);
    float *Md, *Nd, *Pd;

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, k*n*sizeof(float));
    hipMemcpy(Md,M,k*n*sizeof(float),hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, n*m*sizeof(float));
    hipMemcpy(Nd,N,n*m*sizeof(float),hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,k*m*sizeof(float));

    //Setup the execution configuration
    dim3 dimBlock(k,m);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,n,m);

    //Transfer P from device to host
    hipMemcpy(P,Pd,m*k*sizeof(float),hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}

